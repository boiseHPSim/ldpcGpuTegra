
#include "CTimer.h"


CTimer::CTimer(bool _start_timer_){
	hipEventCreate(&_start);
	hipEventCreate(&_stop);

    if(_start_timer_ == true){
    	hipEventRecord(_start, 0);
        isRunning = true;
    }else{
        isRunning = false;
    }
}


CTimer::CTimer(){
	hipEventCreate(&_start);
	hipEventCreate(&_stop);
	isRunning = false;
}


CTimer::~CTimer(){
	hipEventDestroy(_start);
	hipEventDestroy(_stop);
}


void CTimer::start(){
    if( isRunning == true )
	{
        cout << "(EE) CTimer :: trying to start a CTimer object that is already running !" << endl;
    }
    else
	{
        isRunning = true;
    	hipEventRecord(_start, 0);
    }
}


void CTimer::stop(){
	cout << "CTimer::stop()" << endl;
    if( isRunning == false ){
        cout << "(EE) CTimer :: trying to stop a CTimer object that is not running !" << endl;
    }else{
        hipEventRecord(_stop, 0);
        isRunning = false;
    }
}


void CTimer::reset(){
	hipEventRecord(_start, 0);
}


long CTimer::get_time_ns(){
	float elapsedTime;
	if( isRunning == true ){
	    hipEventRecord(_stop, 0);
	    hipEventSynchronize(_stop);
	}
	hipEventElapsedTime(&elapsedTime, _start, _stop); // that's our time!
	return (long)(1000.0 * 1000.0 * elapsedTime);
}


long CTimer::get_time_us(){
	float elapsedTime;
	if( isRunning == true ){
	    hipEventRecord(_stop, 0);
	    hipEventSynchronize(_stop);
	}
	hipEventElapsedTime(&elapsedTime, _start, _stop); // that's our time!
	return (long)(1000.0 * elapsedTime);
}


long CTimer::get_time_ms(){
	float elapsedTime;
	if( isRunning == true ){
	    hipEventRecord(_stop, 0);
	    hipEventSynchronize(_stop);
	}
	hipEventElapsedTime(&elapsedTime, _start, _stop); // that's our time!
	return (long)(elapsedTime);
}


long CTimer::get_time_sec(){
	return (long)(get_time_ms() / 1000.0);
}
