#include "hip/hip_runtime.h"

#include "GPU_Transpose.h"
#include "simd_functions.h"

__global__ void transposeDiagonal(float *odata, float *idata, int width, int height) {
	__shared__ float tile[TILE_DIM][TILE_DIM + 1];
	int blockIdx_x, blockIdx_y;

	// diagonal reordering
	if (width == height) {
		blockIdx_y = blockIdx.x;
		blockIdx_x = (blockIdx.x + blockIdx.y) % gridDim.x;
	} else {
		int bid = blockIdx.x + gridDim.x * blockIdx.y;
		blockIdx_y = bid % gridDim.y;
		blockIdx_x = ((bid / gridDim.y) + blockIdx_y) % gridDim.x;
	}

	int xIndex = blockIdx_x * TILE_DIM + threadIdx.x;
	int yIndex = blockIdx_y * TILE_DIM + threadIdx.y;
	int index_in = xIndex + (yIndex) * width;
	xIndex = blockIdx_y * TILE_DIM + threadIdx.x;
	yIndex = blockIdx_x * TILE_DIM + threadIdx.y;
	int index_out = xIndex + (yIndex) * height;

	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		tile[threadIdx.y + i][threadIdx.x] = idata[index_in + i * width];
	}

	__syncthreads();

	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		odata[index_out + i * height] = tile[threadIdx.x][threadIdx.y + i];
	}
}


__global__ void transposeDiagonal_and_hard_decision(float *odata, float *idata, int width, int height) {
	__shared__ float tile[TILE_DIM][TILE_DIM + 1];
	int blockIdx_x, blockIdx_y;

	// diagonal reordering
	if (width == height) {
		blockIdx_y = blockIdx.x;
		blockIdx_x = (blockIdx.x + blockIdx.y) % gridDim.x;
	} else {
		int bid = blockIdx.x + gridDim.x * blockIdx.y;
		blockIdx_y = bid % gridDim.y;
		blockIdx_x = ((bid / gridDim.y) + blockIdx_y) % gridDim.x;
	}

	int xIndex    = blockIdx_x * TILE_DIM + threadIdx.x;
	int yIndex    = blockIdx_y * TILE_DIM + threadIdx.y;
	int index_in  = xIndex + (yIndex) * width;
	xIndex        = blockIdx_y * TILE_DIM + threadIdx.x;
	yIndex        = blockIdx_x * TILE_DIM + threadIdx.y;
	int index_out = xIndex + (yIndex) * height;

	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		float data = idata[index_in + i * width];
		union{ float f; unsigned int u; } resu; resu.u = data > 0;
		tile[threadIdx.y + i][threadIdx.x] = resu.f;
	}

	__syncthreads();

	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		odata[index_out + i * height] = tile[threadIdx.x][threadIdx.y + i];
	}
}

__global__ void transposeDiagonal_and_hard_decision(unsigned int* _odata, unsigned int* _idata, int width, int height) {
	float *odata = (float*)_odata;
	float *idata = (float*)_idata;
	__shared__ float tile[TILE_DIM][TILE_DIM + 1];
	int blockIdx_x, blockIdx_y;

	// diagonal reordering
	if (width == height) {
		blockIdx_y = blockIdx.x;
		blockIdx_x = (blockIdx.x + blockIdx.y) % gridDim.x;
	} else {
		int bid = blockIdx.x + gridDim.x * blockIdx.y;
		blockIdx_y = bid % gridDim.y;
		blockIdx_x = ((bid / gridDim.y) + blockIdx_y) % gridDim.x;
	}

	int xIndex   = blockIdx_x * TILE_DIM + threadIdx.x;
	int yIndex   = blockIdx_y * TILE_DIM + threadIdx.y;
	int index_in = xIndex + (yIndex) * width;
	xIndex       = blockIdx_y * TILE_DIM + threadIdx.x;
	yIndex       = blockIdx_x * TILE_DIM + threadIdx.y;
	int index_out = xIndex + (yIndex) * height;

	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		union{ float f; unsigned int u; } data;
		data.f = idata[index_in + i * width];			//
		data.u = vsetgts4(data.u, 0x00000000);			// HARD DECISION HERE ...
		tile[threadIdx.y + i][threadIdx.x] = data.f;	//
	}

	__syncthreads();

	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		odata[index_out + i * height] = tile[threadIdx.x][threadIdx.y + i];
	}
}
