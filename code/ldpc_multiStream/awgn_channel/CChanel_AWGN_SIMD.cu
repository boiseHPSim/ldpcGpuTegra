#include "hip/hip_runtime.h"
#include "CChanel_AWGN_SIMD.h"

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
      printf("Error (%d) at %s:%d\n", x, __FILE__,__LINE__);            \
      exit(0);}} while(0)

__global__ void GenerateNoiseAndTransform(const float *A, const float *B, int *C, float SigB, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N)
    {
		float vSin, vCos, x, y;
	    union {char c[4]; unsigned int i;} res_a = {0, 0, 0, 0};
	    union {char c[4]; unsigned int i;} res_b = {0, 0, 0, 0};

		for(int p=0; p<4; p++){
			x  = sqrt(-2.0 * log( A[i + p * N] ));
			y  = B[i + p * N];
			sincosf(_2pi * y, &vSin, &vCos);
			float v1   = (-1.0 + (x * vSin) * SigB);
			float v2   = (-1.0 + (x * vCos) * SigB);
			res_a.c[p] = (char)fminf( fmaxf(8.0f * v1, -31.0f), 31.0f);
			res_b.c[p] = (char)fminf( fmaxf(8.0f * v2, -31.0f), 31.0f);
		}
        C[i]   = res_a.i;
        C[i+N] = res_b.i;

    }
}

//#define SEQ_LEVEL 1

CChanel_AWGN_SIMD::CChanel_AWGN_SIMD(CTrame *t, int _BITS_LLR, bool QPSK, bool Es_N0) : CChanel(t, _BITS_LLR, QPSK, Es_N0){
	hiprandStatus_t Status;

	SEQ_LEVEL = 1 + ((_data > 10000) ? 3 : 0);

	unsigned int nb_ech = (_frames * _data) / SEQ_LEVEL;
	Status = hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
	CURAND_CALL(Status);
    Status = hiprandSetPseudoRandomGeneratorSeed(generator, 1234ULL);
	CURAND_CALL(Status);
	CUDA_MALLOC_DEVICE(&device_A, nb_ech/2,__FILE__, __LINE__);
    CUDA_MALLOC_DEVICE(&device_B, nb_ech/2,__FILE__, __LINE__);
    CUDA_MALLOC_DEVICE(&device_R, nb_ech  ,__FILE__, __LINE__);
}

CChanel_AWGN_SIMD::~CChanel_AWGN_SIMD(){
	hipError_t Status;
	Status = hipFree(device_A);
	ERROR_CHECK(Status, (char*)__FILE__, __LINE__);
	Status = hipFree(device_B);
	ERROR_CHECK(Status, (char*)__FILE__, __LINE__);
	Status = hipFree(device_R);
	ERROR_CHECK(Status, (char*)__FILE__, __LINE__);
	hiprandStatus_t eStatus;
    eStatus = hiprandDestroyGenerator(generator);
	CURAND_CALL(eStatus);
//    printf("(DD) STOP  CChanel_AWGN_SIMD::~CChanel_AWGN_SIMD() !\n");
}

void CChanel_AWGN_SIMD::configure(double _Eb_N0) 
{
    rendement = (float) (_vars) / (float) (_data);
    if (es_n0) {
        Eb_N0 = _Eb_N0 - 10.0 * log10(2 * rendement);
    } else {
        Eb_N0 = _Eb_N0;
    }
    double interm = 10.0 * log10(rendement);
    interm        = -0.1*((double)Eb_N0+interm);
    SigB          = sqrt(pow(10.0,interm)/2);
}

#include <limits.h>
#define MAX_RANDOM LONG_MAX    /* Maximum value of random() */


double CChanel_AWGN_SIMD::awgn(double amp)
{
    return 0.00;
}

#define QPSK 0.707106781
#define BPSK 1.0


void CChanel_AWGN_SIMD::generate()
{
	size_t nb_rand_data = _frames*_data / 2 / SEQ_LEVEL;
	CURAND_CALL( hiprandGenerateUniform( generator, device_A, nb_rand_data ) );
	CURAND_CALL( hiprandGenerateUniform( generator, device_B, nb_rand_data ) );

	for(int i=0; i<4 * SEQ_LEVEL; i++){

		// GENERATION DE DU BRUIT DU CANAL
		size_t nb_noise_sample = nb_rand_data / 2;
		int threadsPerBlock = 1024;
		size_t blocksPerGrid   = (nb_noise_sample  + threadsPerBlock - 1) / threadsPerBlock;
		GenerateNoiseAndTransform<<<blocksPerGrid, threadsPerBlock>>>(device_A, device_B, (int*)device_R, (float)SigB, nb_noise_sample/2);

		hipError_t eStatus = hipMemcpyAsync(&t_noise_data[i * nb_noise_sample], device_R, nb_noise_sample * sizeof(float), hipMemcpyDeviceToHost);

		if( i != 3 ){
		CURAND_CALL( hiprandGenerateUniform( generator, device_A, nb_rand_data ) );
		CURAND_CALL( hiprandGenerateUniform( generator, device_B, nb_rand_data ) );
		}
		hipDeviceSynchronize();
		ERROR_CHECK(hipGetLastError(), __FILE__, __LINE__);

	}
}
